#include "NeuralNetGPU.cuh"
#include <iostream>

void NeuralNetGPU::safeMalloc(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda malloc failed..." << std::endl;
	}
}

void NeuralNetGPU::safeCopy(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda copy failed..." << std::endl;
	}
}

void NeuralNetGPU::forwardProp(std::vector<double>& inputValues)
{
	double* host_inputValues = new double[inputValues.size()];
	double* devi_inputValues = nullptr;

	this->safeMalloc(
		hipMalloc(&devi_inputValues, sizeof(double) * inputValues.size())
	);
	this->safeCopy(
		hipMemcpy(
			devi_inputValues,
			&inputValues[0],
			sizeof(double) * inputValues.size(),
			hipMemcpyHostToDevice
		)
	);

	cudaForwardProp<<<1, 1>>>(devi_inputValues);

	this->safeCopy(
		hipMemcpy(
			host_inputValues,
			devi_inputValues,
			sizeof(double) * inputValues.size(),
			hipMemcpyDeviceToHost
		)
	);

	std::cout << "GPU ANSWER " << host_inputValues[0] << std::endl;

	delete[] host_inputValues;

	hipFree(devi_inputValues);
}

void NeuralNetGPU::release()
{
	hipDeviceReset();
}
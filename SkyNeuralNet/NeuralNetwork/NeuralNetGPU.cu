#include "NeuralNetGPU.cuh"
#include <iostream>
#include "ActivationFunction.h"

void NeuralNetGPU::safeMalloc(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda malloc failed..." << std::endl;
	}
}

void NeuralNetGPU::safeCopy(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda copy failed..." << std::endl;
	}
}

NeuralNetGPU::NeuralNetGPU()
	: numNeurons(0), host_neuronOutputs(nullptr), devi_neuronOutputs(nullptr)
{
}

NeuralNetGPU::~NeuralNetGPU() { }

void NeuralNetGPU::setupTrainingSession(
	std::vector<Layer*>& layers,
	const unsigned int numNeurons,
	const unsigned int numWeights,
	const unsigned int maxNumNeuronsInLayer
)
{
	this->numLayers = layers.size();
	this->numNeurons = numNeurons;
	this->numWeights = numWeights;
	this->maxNumNeuronsInLayer = maxNumNeuronsInLayer;


	// ----- Variables for CPU <-> GPU communication -----

	// All output values
	this->host_neuronOutputs = new double[this->numNeurons];
	this->devi_neuronOutputs = nullptr;

	// All weights
	this->host_neuronWeights = new double[this->numWeights];
	this->devi_neuronWeights = nullptr;

	// Number of neurons per layer
	this->host_neuronsPerLayer = new int[this->numLayers];
	this->devi_neuronsPerLayer = nullptr;

	// Insert number of neurons per layer
	for(int i = 0; i < this->numLayers; ++i)
		host_neuronsPerLayer[i] = layers[i]->getNeurons().size();

	// Allocate variables on GPU
	this->safeMalloc(
		hipMalloc(&devi_neuronOutputs, sizeof(double) * this->numNeurons)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronWeights, sizeof(double) * this->numWeights)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronsPerLayer, sizeof(int) * this->numLayers)
	);

	// Copy over number of neurons per layer, 
	// since this stays static
	this->safeCopy(
		hipMemcpy(
			devi_neuronsPerLayer,
			host_neuronsPerLayer,
			sizeof(int) * layers.size(),
			hipMemcpyHostToDevice
		)
	);
}

void NeuralNetGPU::forwardProp(std::vector<Layer*>& layers)
{
	// Neuron outputs, weights
	unsigned int currentNeuronIndex = 0;
	unsigned int currentWeightIndex = 0;
	for (int i = 0; i < layers.size(); ++i)
	{
		std::vector<Neuron*>& layerNeurons = layers[i]->getNeurons();

		// Loop through neurons
		for (int j = 0; j < layerNeurons.size(); ++j)
		{
			// Insert output values 
			// (only the first layer actually matters here)
			host_neuronOutputs[currentNeuronIndex++] = layerNeurons[j]->getOutputValue();
		
			// Loop through weights
			std::vector<double>& currentWeights = layerNeurons[j]->getWeights();
			for (int k = 0; k < currentWeights.size(); ++k)
			{
				// Insert weights
				host_neuronWeights[currentWeightIndex++] = currentWeights[k];
			}
		}
	}
	this->safeCopy(
		hipMemcpy(
			devi_neuronOutputs,
			host_neuronOutputs,
			sizeof(double) * numNeurons,
			hipMemcpyHostToDevice
		)
	);
	this->safeCopy(
		hipMemcpy(
			devi_neuronWeights,
			host_neuronWeights,
			sizeof(double) * numWeights,
			hipMemcpyHostToDevice
		)
	);

	// ----- Execute on GPU -----
	cudaForwardProp<<<1, maxNumNeuronsInLayer >>>(
		devi_neuronOutputs, 
		devi_neuronWeights,
		devi_neuronsPerLayer,
		(int) this->numLayers
	);
	hipDeviceSynchronize();

	// Extract results
	this->safeCopy(
		hipMemcpy(
			host_neuronOutputs,
			devi_neuronOutputs,
			sizeof(double) * numNeurons,
			hipMemcpyDeviceToHost
		)
	);

	// Let the CPU calculate activation function for output layer
	std::vector<Neuron*>& lastLayerNeurons = layers.back()->getNeurons();
	for (int i = 0; i < lastLayerNeurons.size(); ++i)
	{
		int currentIndex = this->numNeurons - lastLayerNeurons.size() + i;

		host_neuronOutputs[currentIndex] = 
			ActivationFunction::activateOutput(host_neuronOutputs[currentIndex]);
	}


	// ----- Apply results to network -----
	unsigned int currentNeuronStride = layers[0]->getNeurons().size();
	for (int i = 1; i < layers.size(); ++i)
	{
		// Set
		layers[i]->setAllOutputs(&host_neuronOutputs[currentNeuronStride]);

		// Move stride
		currentNeuronStride += layers[i]->getNeurons().size();
	}
}

void NeuralNetGPU::releaseTrainingSession()
{
	delete[] host_neuronOutputs;
	delete[] host_neuronWeights;
	delete[] host_neuronsPerLayer;

	hipFree(devi_neuronOutputs);
	hipFree(devi_neuronWeights);
	hipFree(devi_neuronsPerLayer);

	hipDeviceReset();
}
#include "hip/hip_runtime.h"
#include "NeuralNetGPU.cuh"
#include <iostream>
#include "ActivationFunction.h"

void NeuralNetGPU::safeMalloc(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda malloc failed..." << std::endl;
	}
}

void NeuralNetGPU::safeCopy(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda copy failed..." << std::endl;
	}
}

NeuralNetGPU::NeuralNetGPU()
	: numLayers(0), numNeurons(0), numWeights(0), maxNumNeuronsInLayer(0),
	host_neuronOutputs(nullptr), devi_neuronOutputs(nullptr),
	host_neuronWeights(nullptr), devi_neuronWeights(nullptr),
	host_neuronsPerLayer(nullptr), devi_neuronsPerLayer(nullptr),

	host_neuronGradients(nullptr),
	devi_neuronGradients(nullptr),
	host_neuronDeltaWeights(nullptr),
	devi_neuronDeltaWeights(nullptr)
{
}

NeuralNetGPU::~NeuralNetGPU() { }

void NeuralNetGPU::setupTrainingSession(
	std::vector<Layer*>& layers,
	const unsigned int numNeurons,
	const unsigned int numWeights,
	const unsigned int maxNumNeuronsInLayer
)
{
	this->numLayers = layers.size();
	this->numNeurons = numNeurons;
	this->numWeights = numWeights;
	this->maxNumNeuronsInLayer = maxNumNeuronsInLayer;


	// ----- Variables for CPU <-> GPU communication -----

	// All output values
	this->host_neuronOutputs = new double[this->numNeurons]{ };
	this->devi_neuronOutputs = nullptr;

	// All weights
	this->host_neuronWeights = new double[this->numWeights]{ };
	this->devi_neuronWeights = nullptr;

	// Insert initial weights
	unsigned int currentWeightIndex = 0;
	for (int i = 0; i < layers.size(); ++i)
	{
		// Loop through neurons
		std::vector<Neuron*>& layerNeurons = layers[i]->getNeurons();
		for (int j = 0; j < layerNeurons.size(); ++j)
		{
			// Loop through weights
			std::vector<double>& currentWeights = layerNeurons[j]->getWeights();
			for (int k = 0; k < currentWeights.size(); ++k)
			{
				// Insert weights
				this->host_neuronWeights[currentWeightIndex++] = currentWeights[k];
			}
		}
	}

	// Number of neurons per layer
	this->host_neuronsPerLayer = new int[this->numLayers]{ };
	this->devi_neuronsPerLayer = nullptr;

	// Insert number of neurons per layer
	for(int i = 0; i < this->numLayers; ++i)
		host_neuronsPerLayer[i] = layers[i]->getNeurons().size();


	// All gradients
	this->host_neuronGradients = new double[this->numNeurons]{ };
	this->devi_neuronGradients = nullptr;

	// All delta weights
	this->host_neuronDeltaWeights = new double[this->numWeights]{ };
	this->devi_neuronDeltaWeights = nullptr;


	// Allocate variables on GPU
	this->safeMalloc(
		hipMalloc(&devi_neuronOutputs, sizeof(double) * this->numNeurons)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronWeights, sizeof(double) * this->numWeights)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronsPerLayer, sizeof(int) * this->numLayers)
	);


	this->safeMalloc(
		hipMalloc(&devi_neuronGradients, sizeof(double) * this->numNeurons)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronDeltaWeights, sizeof(double) * this->numWeights)
	);


	// Copy over number of neurons per layer, 
	// since this stays static
	this->safeCopy(
		hipMemcpy(
			devi_neuronsPerLayer,
			host_neuronsPerLayer,
			sizeof(int) * layers.size(),
			hipMemcpyHostToDevice
		)
	);

	// Copy over weights once, then keep them on the GPU
	this->safeCopy(
		hipMemcpy(
			this->devi_neuronWeights,
			this->host_neuronWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyHostToDevice
		)
	);
}

void NeuralNetGPU::forwardProp(
	std::vector<Layer*>& layers, 
	const std::vector<double>& inputValues
)
{
	// Get neuron outputs from input layer
	memcpy(this->host_neuronOutputs, &inputValues[0], sizeof(double) * inputValues.size());

	this->safeCopy(
		hipMemcpy(
			this->devi_neuronOutputs,
			this->host_neuronOutputs,
			sizeof(double) * this->numNeurons,
			hipMemcpyHostToDevice
		)
	);

	// ----- Execute on GPU -----
	cudaForwardProp<<<1, this->maxNumNeuronsInLayer>>>(
		this->devi_neuronOutputs, 
		this->devi_neuronWeights,
		this->devi_neuronsPerLayer,
		(int) this->numLayers
	);
	hipDeviceSynchronize();

	// Extract results
	this->safeCopy(
		hipMemcpy(
			this->host_neuronOutputs,
			this->devi_neuronOutputs,
			sizeof(double) * this->numNeurons,
			hipMemcpyDeviceToHost
		)
	);

	// Let the CPU calculate activation function for output layer.
	// This is to keep precision when using std::exp().
	std::vector<Neuron*>& lastLayerNeurons = layers.back()->getNeurons();
	int currentIndex = this->numNeurons - lastLayerNeurons.size();
	for (int i = 0; i < lastLayerNeurons.size(); ++i)
	{
		host_neuronOutputs[currentIndex++] = 
			ActivationFunction::activateOutput(host_neuronOutputs[currentIndex]);
	}

	// ----- Apply output results to network -----
	unsigned int currentNeuronStride = this->numNeurons - layers.back()->getNeurons().size();
	layers.back()->setAllOutputs(&host_neuronOutputs[currentNeuronStride]);
}

void NeuralNetGPU::backProp(std::vector<Layer*>& layers)
{
	// Get gradients from output layer,
	// since these are calculated on the CPU
	std::vector<Neuron*>& currentNeurons = layers.back()->getNeurons();
	unsigned int numLastNeurons = currentNeurons.size();
	unsigned int neuronIndex = this->numNeurons - numLastNeurons;
	for (int j = 0; j < currentNeurons.size(); ++j)
	{
		// Insert gradients
		this->host_neuronGradients[neuronIndex++] = currentNeurons[j]->getGradient();
	}

	this->safeCopy(
		hipMemcpy(
			this->devi_neuronGradients,
			this->host_neuronGradients,
			sizeof(double) * this->numNeurons,
			hipMemcpyHostToDevice
		)
	);

	// ----- Execute on GPU -----
	cudaBackProp<<<1, this->maxNumNeuronsInLayer>>>(
		this->devi_neuronOutputs,
		this->devi_neuronWeights, 
		this->devi_neuronDeltaWeights,
		this->devi_neuronGradients,
		this->devi_neuronsPerLayer,
		(int) this->numLayers,
		Neuron::getETA(),
		Neuron::getALPHA()
	);
	hipDeviceSynchronize();
}

void NeuralNetGPU::extractApplyResults(std::vector<Layer*>& layers)
{
	// Extract results

	// Gradients
	this->safeCopy(
		hipMemcpy(
			this->host_neuronGradients,
			this->devi_neuronGradients,
			sizeof(double) * this->numNeurons,
			hipMemcpyDeviceToHost
		)
	);
	// Weights
	this->safeCopy(
		hipMemcpy(
			this->host_neuronWeights,
			this->devi_neuronWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyDeviceToHost
		)
	);
	// Delta weights
	this->safeCopy(
		hipMemcpy(
			this->host_neuronDeltaWeights,
			this->devi_neuronDeltaWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyDeviceToHost
		)
	);

	// Apply results to network
	// Gradients, weights, delta weights
	unsigned int neuronIndex = 0;
	unsigned int weightIndex = 0;
	for (int i = 0; i < layers.size(); ++i)
	{
		std::vector<Neuron*>& currentNeurons = layers[i]->getNeurons();

		for (int j = 0; j < currentNeurons.size(); ++j)
		{
			// Set gradient
			currentNeurons[j]->setGradient(
				this->host_neuronGradients[neuronIndex++]
			);

			// Set weights and delta weights
			for (int k = 0; k < currentNeurons[j]->getWeights().size(); ++k)
			{
				currentNeurons[j]->setWeight(
					k,
					this->host_neuronWeights[weightIndex],
					this->host_neuronDeltaWeights[weightIndex]
				);

				weightIndex++;
			}
		}
	}
}

void NeuralNetGPU::releaseTrainingSession()
{
	delete[] this->host_neuronOutputs;
	delete[] this->host_neuronWeights;
	delete[] this->host_neuronsPerLayer;

	delete[] this->host_neuronGradients;
	delete[] this->host_neuronDeltaWeights;

	hipFree(this->devi_neuronWeights);
	hipFree(this->devi_neuronOutputs);
	hipFree(this->devi_neuronsPerLayer);

	hipFree(this->devi_neuronGradients);
	hipFree(this->devi_neuronDeltaWeights);

	hipDeviceReset();
}
#include "NeuralNetGPU.cuh"
#include <iostream>
#include "ActivationFunction.h"

void NeuralNetGPU::safeMalloc(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda malloc failed..." << std::endl;
	}
}

void NeuralNetGPU::safeCopy(const hipError_t& error)
{
	if (error != hipSuccess)
	{
		std::cout << "Cuda copy failed..." << std::endl;
	}
}

NeuralNetGPU::NeuralNetGPU()
	: numLayers(0), numNeurons(0), numWeights(0), maxNumNeuronsInLayer(0),
	host_neuronOutputs(nullptr), devi_neuronOutputs(nullptr),
	host_neuronWeights(nullptr), devi_neuronWeights(nullptr),
	host_neuronsPerLayer(nullptr), devi_neuronsPerLayer(nullptr),

	host_neuronGradients(nullptr),
	devi_neuronGradients(nullptr),
	host_neuronDeltaWeights(nullptr),
	devi_neuronDeltaWeights(nullptr)
{
}

NeuralNetGPU::~NeuralNetGPU() { }

void NeuralNetGPU::setupTrainingSession(
	std::vector<Layer*>& layers,
	const unsigned int numNeurons,
	const unsigned int numWeights,
	const unsigned int maxNumNeuronsInLayer
)
{
	this->numLayers = layers.size();
	this->numNeurons = numNeurons;
	this->numWeights = numWeights;
	this->maxNumNeuronsInLayer = maxNumNeuronsInLayer;


	// ----- Variables for CPU <-> GPU communication -----

	// All output values
	this->host_neuronOutputs = new double[this->numNeurons];
	this->devi_neuronOutputs = nullptr;

	// All weights
	this->host_neuronWeights = new double[this->numWeights];
	this->devi_neuronWeights = nullptr;

	// Number of neurons per layer
	this->host_neuronsPerLayer = new int[this->numLayers];
	this->devi_neuronsPerLayer = nullptr;

	// Insert number of neurons per layer
	for(int i = 0; i < this->numLayers; ++i)
		host_neuronsPerLayer[i] = layers[i]->getNeurons().size();

	// All gradients
	this->host_neuronGradients = new double[this->numNeurons];
	this->devi_neuronGradients = nullptr;

	// All delta weights
	this->host_neuronDeltaWeights = new double[this->numWeights];
	this->devi_neuronDeltaWeights = nullptr;

	// Allocate variables on GPU
	this->safeMalloc(
		hipMalloc(&devi_neuronOutputs, sizeof(double) * this->numNeurons)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronWeights, sizeof(double) * this->numWeights)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronsPerLayer, sizeof(int) * this->numLayers)
	);

	this->safeMalloc(
		hipMalloc(&devi_neuronGradients, sizeof(double) * this->numNeurons)
	);
	this->safeMalloc(
		hipMalloc(&devi_neuronDeltaWeights, sizeof(double) * this->numWeights)
	);

	// Copy over number of neurons per layer, 
	// since this stays static
	this->safeCopy(
		hipMemcpy(
			devi_neuronsPerLayer,
			host_neuronsPerLayer,
			sizeof(int) * layers.size(),
			hipMemcpyHostToDevice
		)
	);
}

void NeuralNetGPU::forwardProp(std::vector<Layer*>& layers)
{
	// Get neuron outputs, weights
	unsigned int currentNeuronIndex = 0;
	unsigned int currentWeightIndex = 0;
	for (int i = 0; i < layers.size(); ++i)
	{
		std::vector<Neuron*>& layerNeurons = layers[i]->getNeurons();

		// Loop through neurons
		for (int j = 0; j < layerNeurons.size(); ++j)
		{
			// Insert output values 
			// (only the first layer actually matters here)
			this->host_neuronOutputs[currentNeuronIndex++] = layerNeurons[j]->getOutputValue();
		
			// Loop through weights
			std::vector<double>& currentWeights = layerNeurons[j]->getWeights();
			for (int k = 0; k < currentWeights.size(); ++k)
			{
				// Insert weights
				this->host_neuronWeights[currentWeightIndex++] = currentWeights[k];
			}
		}
	}
	this->safeCopy(
		hipMemcpy(
			this->devi_neuronOutputs,
			this->host_neuronOutputs,
			sizeof(double) * this->numNeurons,
			hipMemcpyHostToDevice
		)
	);
	this->safeCopy(
		hipMemcpy(
			this->devi_neuronWeights,
			this->host_neuronWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyHostToDevice
		)
	);

	// ----- Execute on GPU -----
	cudaForwardProp<<<1, this->maxNumNeuronsInLayer>>>(
		this->devi_neuronOutputs, 
		this->devi_neuronWeights,
		this->devi_neuronsPerLayer,
		(int) this->numLayers
	);
	hipDeviceSynchronize();

	// Extract results
	this->safeCopy(
		hipMemcpy(
			this->host_neuronOutputs,
			this->devi_neuronOutputs,
			sizeof(double) * this->numNeurons,
			hipMemcpyDeviceToHost
		)
	);

	// Let the CPU calculate activation function for output layer
	std::vector<Neuron*>& lastLayerNeurons = layers.back()->getNeurons();
	for (int i = 0; i < lastLayerNeurons.size(); ++i)
	{
		int currentIndex = this->numNeurons - lastLayerNeurons.size() + i;

		host_neuronOutputs[currentIndex] = 
			ActivationFunction::activateOutput(host_neuronOutputs[currentIndex]);
	}


	// ----- Apply results to network -----
	unsigned int currentNeuronStride = layers[0]->getNeurons().size();
	for (int i = 1; i < layers.size(); ++i)
	{
		// Set
		layers[i]->setAllOutputs(&host_neuronOutputs[currentNeuronStride]);

		// Move stride
		currentNeuronStride += layers[i]->getNeurons().size();
	}
}

void NeuralNetGPU::backProp(
	std::vector<Layer*>& layers,
	const std::vector<double>& expectedValues
)
{
	// Get gradients
	unsigned int neuronIndex = 0;
	unsigned int weightIndex = 0;
	for (int i = 0; i < layers.size(); ++i)
	{
		// Loop through neurons
		std::vector<Neuron*>& currentNeurons = layers[i]->getNeurons();
		for (int j = 0; j < currentNeurons.size(); ++j)
		{
			// Insert gradients
			this->host_neuronGradients[neuronIndex++] = currentNeurons[j]->getGradient();

			// Loop through delta weights
			std::vector<double>& currentDeltaWeights = currentNeurons[j]->getDeltaWeights();
			for (int k = 0; k < currentDeltaWeights.size(); ++k)
			{
				// Insert delta weight
				this->host_neuronDeltaWeights[weightIndex++] = currentDeltaWeights[k];
			}
		}
	}
	this->safeCopy(
		hipMemcpy(
			this->devi_neuronGradients,
			this->host_neuronGradients,
			sizeof(double) * this->numNeurons,
			hipMemcpyHostToDevice
		)
	);
	this->safeCopy(
		hipMemcpy(
			this->devi_neuronDeltaWeights,
			this->host_neuronDeltaWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyHostToDevice
		)
	);

	// ----- Execute on GPU -----
	cudaBackProp<<<1, this->maxNumNeuronsInLayer>>>(
		// This is assuming outputs and weights
		// doesn't change before back prop
		this->devi_neuronOutputs,
		this->devi_neuronWeights, 
		this->devi_neuronDeltaWeights,
		this->devi_neuronGradients,
		this->devi_neuronsPerLayer,
		(int) this->numLayers,
		Neuron::getETA(),
		Neuron::getALPHA()
	);
	hipDeviceSynchronize();

	// Extract results

	// Gradients
	this->safeCopy(
		hipMemcpy(
			this->host_neuronGradients,
			this->devi_neuronGradients,
			sizeof(double) * this->numNeurons,
			hipMemcpyDeviceToHost
		)
	);
	// Weights
	this->safeCopy(
		hipMemcpy(
			this->host_neuronWeights,
			this->devi_neuronWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyDeviceToHost
		)
	);
	// Delta weights
	this->safeCopy(
		hipMemcpy(
			this->host_neuronDeltaWeights,
			this->devi_neuronDeltaWeights,
			sizeof(double) * this->numWeights,
			hipMemcpyDeviceToHost
		)
	);

	// ----- Apply results to network -----
	neuronIndex = 0;
	weightIndex = 0;
	for (int i = 0; i < layers.size(); ++i)
	{
		std::vector<Neuron*>& currentNeurons = layers[i]->getNeurons();

		for (int j = 0; j < currentNeurons.size(); ++j)
		{
			// Set gradient
			currentNeurons[j]->setGradient(
				this->host_neuronGradients[neuronIndex++]
			);

			// Set weights and delta weights
			for (int k = 0; k < currentNeurons[j]->getWeights().size(); ++k)
			{
				currentNeurons[j]->setWeight(
					k, 
					this->host_neuronWeights[weightIndex],
					this->host_neuronDeltaWeights[weightIndex]
				);

				weightIndex++;
			}
		}
	}
}

void NeuralNetGPU::releaseTrainingSession()
{
	delete[] this->host_neuronOutputs;
	delete[] this->host_neuronWeights;
	delete[] this->host_neuronsPerLayer;

	delete[] this->host_neuronGradients;
	delete[] this->host_neuronDeltaWeights;

	hipFree(this->devi_neuronWeights);
	hipFree(this->devi_neuronOutputs);
	hipFree(this->devi_neuronsPerLayer);

	hipFree(this->devi_neuronGradients);
	hipFree(this->devi_neuronDeltaWeights);

	hipDeviceReset();
}
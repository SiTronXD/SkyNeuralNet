#include "hip/hip_runtime.h"
﻿#include "NeuralNetGPUFunctions.cuh"

__device__ double activationFunctionHidden(double x)
{
	// Relu
	return fmax(0.0, x);
}
__device__ double activationFunctionDerivativeHidden(double x)
{
	return x >= 0.0 ? 1.0 : 0.0;
}

__device__ double activationFunctionOutput(double x)
{
	// Sigmoid

	// As expected, exp() gives slightly different 
	// results when comparing CUDA exp() and std::exp()
	return 1.0 / (1.0 + exp(-x));
}
__device__ double activationFunctionDerivativeOutput(double x)
{
	double s = activationFunctionOutput(x);

	return s * (1.0 - s);
}

#define MAX_BLOCKING_SIZE 1024

__global__ void cudaForwardProp(
	double* neuronOutputs,
	double* neuronWeights,
	int* neuronsPerLayer,
	int numLayers
)
{
	__shared__ double lastLayerOutputs[MAX_BLOCKING_SIZE];

	int id = blockIdx.x * blockDim.x + threadIdx.x;

	int layerIndexStride = 0;
	int lastLayerIndexStride = 0;
	int lastLayerWeightStride = 0;

	// Go through each layer
	for (int l = 1; l < numLayers; ++l)
	{
		layerIndexStride += neuronsPerLayer[l - 1];

		// Load last layer output values
		// into shared memory
		if (id < neuronsPerLayer[l - 1])
		{
			lastLayerOutputs[id] = 
				neuronOutputs[lastLayerIndexStride + id];
		}
		__syncthreads();

		// Don't calculate output for bias neurons
		if (id < neuronsPerLayer[l] - 1)
		{
			neuronOutputs[layerIndexStride + id] = 0;

			// Go through each neuron from the last layer
			for (int n = 0; n < neuronsPerLayer[l - 1]; ++n)
			{
				double outVal = lastLayerOutputs[n];
				double weightVal = 
					neuronWeights[
						lastLayerWeightStride +
						(neuronsPerLayer[l] - 1) * n + // Ignore bias neuron
						id
					];

				neuronOutputs[layerIndexStride + id] += outVal * weightVal;
			}

			// Activation function for hidden layers
			if (l < numLayers - 1)
			{
				neuronOutputs[layerIndexStride + id] =
					activationFunctionHidden(neuronOutputs[layerIndexStride + id]);
			}
			// Activation function for output layer
			// (Let the CPU do it to keep precision,
			// only takes <100 ms for 5000 training sets)
			/*else
			{
				neuronOutputs[layerIndexStride + id] =
					activationFunctionOutput(neuronOutputs[layerIndexStride + id]);
			}*/
		}
		// Bias neuron
		else if (id == neuronsPerLayer[l] - 1)
		{
			neuronOutputs[layerIndexStride + id] = 1.0;
		}

		lastLayerWeightStride += (neuronsPerLayer[l - 1]) * (neuronsPerLayer[l] - 1);
		lastLayerIndexStride = layerIndexStride;

		__syncthreads();
	}
}

__global__ void cudaCalcGradients(
	double* neuronOutputs,
	double* neuronWeights,
	double* neuronGradients,
	int* neuronsPerLayer,
	int numLayers
)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	// ----- Calculate gradients in hidden layers -----

	// Stride points to last hidden layer
	int layerStride = 0;
	int nextLayerStride = 0;
	for (int i = 0; i < numLayers - 1 - 1; ++i)
		layerStride += neuronsPerLayer[i];
	nextLayerStride = layerStride + neuronsPerLayer[numLayers - 1 - 1];

	// Stride points to last hidden layer weights
	int weightStride = 0;
	for (int i = 0; i < numLayers - 1 - 1; ++i)
	{
		// += <number of neurons> * <number of weights for each neuron>
		weightStride += neuronsPerLayer[i] * (neuronsPerLayer[i + 1] - 1);
	}

	// Go through each hidden layer, back to front, 
	// starting from the last hidden layer
	for (int i = numLayers - 1 - 1; i > 0; --i)
	{
		// Make sure this thread can work
		if (id < neuronsPerLayer[i])
		{
			// Sum weight gradients
			double swg = 0.0;

			for (int j = 0; j < neuronsPerLayer[i + 1] - 1; ++j)
			{
				// += <weight to next neuron> * <next neuron gradient>
				swg +=
					neuronWeights[weightStride + (neuronsPerLayer[i + 1] - 1) * id + j] *
					neuronGradients[nextLayerStride + j];
			}

			neuronGradients[layerStride + id] =
				swg *
				activationFunctionDerivativeHidden(neuronOutputs[layerStride + id]);
		}

		nextLayerStride = layerStride;
		layerStride -= neuronsPerLayer[i - 1];
		weightStride -= neuronsPerLayer[i - 1] * (neuronsPerLayer[i] - 1);

		__syncthreads();
	}

	// ----- Update weights -----

	// Go through all layers, except output layer
	/*nextLayerStride = neuronsPerLayer[0];
	layerStride = 0;
	weightStride = 0;
	for (int i = 0; i < numLayers - 1; ++i)
	{
		// Make sure this thread can work
		if (id < neuronsPerLayer[i])
		{
			// Go through weights
			for (int j = 0; j < neuronsPerLayer[i + 1] - 1; ++j)
			{
				int weightIndex =
					weightStride +
					(neuronsPerLayer[i + 1] - 1) * id + // Ignore bias neuron
					j;

				double oldDeltaWeight = neuronDeltaWeights[weightIndex];
				double newDeltaWeight =
					eta * neuronOutputs[layerStride + id] * neuronGradients[nextLayerStride + j] +
					alpha * oldDeltaWeight;

				// Apply weight and delta weight
				neuronDeltaWeights[weightIndex] = newDeltaWeight;
				neuronWeights[weightIndex] += newDeltaWeight;
			}
		}

		weightStride += neuronsPerLayer[i] * (neuronsPerLayer[i + 1] - 1);
		layerStride = nextLayerStride;
		nextLayerStride += neuronsPerLayer[i + 1];

		__syncthreads();
	}*/
}

__global__ void cudaUpdateWeights(
	double* neuronOutputs,
	double* neuronWeights,
	double* neuronDeltaWeights,
	double* neuronGradients,
	int* thisNeuronIndex,
	int* nextNeuronIndex,
	int numWeights,
	float eta,
	float alpha
)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;

	if (id < numWeights)
	{
		double oldDeltaWeight = neuronDeltaWeights[id];
		double newDeltaWeight =
			eta * neuronOutputs[thisNeuronIndex[id]] * neuronGradients[nextNeuronIndex[id]] +
			alpha * oldDeltaWeight;

		// Apply weight and delta weight
		neuronDeltaWeights[id] = newDeltaWeight;
		neuronWeights[id] += newDeltaWeight;
	}

	__syncthreads();
}
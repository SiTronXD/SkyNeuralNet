#include <iostream>
#include <hip/hip_runtime.h>



using namespace std;

__global__ void AddIntsCUDA(int* a, int* b)
{
	a[0] += b[0];
}

int main()
{
	int a = 5, b = 9;
	int* d_a, * d_b;

	// Allocate memory on gpu
	if (hipMalloc(&d_a, sizeof(int)) != hipSuccess)
	{
		cout << "Error allocating memory!" << endl;
		return 1;
	}
	if (hipMalloc(&d_b, sizeof(int)) != hipSuccess)
	{
		cout << "Error allocating memory!" << endl;
		hipFree(d_a);
		return 1;
	}

	// Copy values into allocated memory
	if (hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
	{
		cout << "Error copying memory!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 1;
	}
	if (hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice) != hipSuccess)
	{
		cout << "Error copying memory!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 1;
	}

	// Execute function on gpu
	AddIntsCUDA <<<1, 1 >>> (d_a, d_b);
	hipDeviceSynchronize();

	// Copy calculated value from device to host
	if (hipMemcpy(&a, d_a, sizeof(int), hipMemcpyDeviceToHost) != hipSuccess)
	{
		cout << "Error copying memory!" << endl;
		hipFree(d_a);
		hipFree(d_b);
		return 1;
	}

	cout << "a: " << a << endl;

	hipFree(d_a);
	hipFree(d_b);

	hipDeviceReset();

	getchar();

	return 0;
}